
// Modified CUDA Add Example
// This example takes 2 float arrays of size 1M and adds them together.
// Prints out Total Runtime.

#include "hip/hip_runtime.h"


#include <stdio.h>


#include <iostream>
#include <iomanip>
#include <chrono>



hipError_t addWithCuda(float *c, const float *a, const float *b, unsigned int size);

__global__ void addKernel(float *c, const float *a, const float *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];

	//printf("ThreadIdx.x : %d " , i );

	//printf("a[i] = %.2f", a[i]);
	//printf("b[i] = %.2f", b[i]);
	//printf("c[i] = %.2f \n", c[i]);
}

int main()
{


	// Instance
    const int arraySize =  1 << 20;  // 1 Million elements
	float *x = new float[arraySize]; // Input 1
	float *y = new float[arraySize]; // Input 2
	float *z = new float[arraySize]; // Output

	// Initialize
	for (int i = 0; i < arraySize; i++)
	{
		x[i] = 1.0f;
		y[i] = 2.0f;
		z[i] = 0.0f;
	}

    //const int a[arraySize] = { 1, 2, 3, 4, 5 };
    //const int b[arraySize] = { 10, 20, 30, 40, 50 };
    //int c[arraySize] = { 0 };

	//Timing Variables
	auto start = std::chrono::high_resolution_clock::now();
	std::ios_base::sync_with_stdio(false);

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(z, x, y, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

	// Calculating total time taken by the program. 
	auto end = std::chrono::high_resolution_clock::now();
	double time_taken = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
	time_taken *= 1e-9;
	std::cout << "Time taken by program: " << std::fixed << time_taken << std::setprecision(9) << " sec. \n" << std::endl;


    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }


	//Print arrays
	printf("{x[0], x[1], x[2], x[3],x[4], ...} + {y[0], y[1], y[2], y[3], y[4], ...} = {%.2f,%.2f,%.2f,%.2f,%.2f, ...}\n",
		z[0], z[1], z[2], z[3], z[4]);

	printf("Complete.\n");


    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(float *c, const float *a, const float *b, unsigned int size)
{
    float *dev_a = 0;
    float *dev_b = 0;
    float *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, 256>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
